#include "hip/hip_runtime.h"
#include "node.h"
#include <stdio.h>

#define BATCH 32

__global__ void JustCopy(uint8_t *bf, uint8_t *res, int cols)
{
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    res[0] = 4;
    bf[0] = 5;
    for(int i=0;i<BATCH*FIELD_Y*FIELD_X;i++){
        bf[i] = 10;
    }
    //if(i < cols) {
    //    bf[i*FIELD_Y] = res[0];
    //}
}

int main(void)
{
    uint8_t start[FIELD_X][FIELD_Y];
    memset(start, 0, sizeof(start));
    start[0][1]=2;
    start[0][3]=2;

    uint8_t batch_fields[BATCH][FIELD_X][FIELD_Y];
    for(int i=0;i<BATCH;i++)
        memcpy(batch_fields[i], start, sizeof(start));

    uint8_t results[BATCH*FIELD_X][FIELD_X][FIELD_Y];
    uint8_t *d_batch_fields;
    uint8_t *d_results;

    hipMalloc(&d_batch_fields, sizeof(batch_fields));
    hipMalloc(&d_results, sizeof(results));
    
    hipMemcpy(d_batch_fields, batch_fields, sizeof(batch_fields), hipMemcpyHostToDevice);

    for(int i=0;i<FIELD_X;i++)
        hipMemcpy(d_results+i*BATCH*FIELD_SZ, d_batch_fields, sizeof(batch_fields), hipMemcpyDeviceToDevice);

    JustCopy<<<3,18>>>(d_batch_fields, d_results, BATCH*FIELD_X);

    uint8_t end[FIELD_X][FIELD_Y];
    //hipMemcpy(end, d_results, sizeof(end), hipMemcpyDeviceToHost);
    hipMemcpy(end, d_batch_fields, sizeof(end), hipMemcpyDeviceToHost);
    print_field(end);

    hipFree(d_batch_fields);
    hipFree(d_results);
    return 0;
}
